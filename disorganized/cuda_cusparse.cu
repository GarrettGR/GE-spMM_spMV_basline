#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#define CHECK_CUDA(func) { \
  hipError_t status = (func); \
  if (status != hipSuccess) { \
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(status)); \
    exit(1); \
  } \
}

#define CHECK_CUSPARSE(func) { \
  hipsparseStatus_t status = (func); \
  if (status != HIPSPARSE_STATUS_SUCCESS) { \
    fprintf(stderr, "CUSPARSE Error: %d\n", status); \
    exit(1); \
  } \
}

void printProfiling(int m, int n, int nnzA, int nnzB, int nnzC, double initTime, double multTime, double totalTime, size_t memUsage) {
  double flops = 2.0 * nnzC;
  double gflops = flops / multTime / 1e9;
  double densityA = 100.0 * nnzA / (m * n);
  double densityB = 100.0 * nnzB / (m * n);
  double densityC = 100.0 * nnzC / (m * n);

  printf("==================\n");
  printf("Profiling Results:\n");
  printf("==================\n");
  printf("Initialization Time: %.4f seconds\n", initTime);
  printf("Multiplication Time: %.4f seconds\n", multTime);
  printf("Total Time: %.4f seconds\n", totalTime);
  printf("Peak Memory Usage: %.2f MB\n", memUsage / (1024.0 * 1024.0));
  printf("Total FLOPS: %.2e\n", flops);
  printf("Performance: %.2f GFLOPS\n", gflops);

  printf("\n");

  printf("==================\n");
  printf("Matrix Statistics:\n");
  printf("==================\n");
  printf("Input Matrix A: %d x %d, %d non-zeros (%.2f%% dense)\n", m, n, nnzA, densityA);
  printf("Input Matrix B: %d x %d, %d non-zeros (%.2f%% dense)\n", n, n, nnzB, densityB);
  printf("Result Matrix C: %d x %d, %d non-zeros (%.2f%% dense)\n", m, n, nnzC, densityC);
}

int main() {
  int m = 10000;
  int n = 10000;
  float sparsity = 0.001;
  int nnzA = sparsity * m * n;
  int nnzB = sparsity * n * n;

  // Initialize matrix data
  int *csrRowPtrA, *csrColIndA;
  float *csrValA;
  int *csrRowPtrB, *csrColIndB;
  float *csrValB;
  int *csrRowPtrC;
  int *csrColIndC;
  float *csrValC;

  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));

  // Allocate host memory
  csrRowPtrA = (int*)malloc((m + 1) * sizeof(int));
  csrColIndA = (int*)malloc(nnzA * sizeof(int));
  csrValA = (float*)malloc(nnzA * sizeof(float));

  csrRowPtrB = (int*)malloc((n + 1) * sizeof(int));
  csrColIndB = (int*)malloc(nnzB * sizeof(int));
  csrValB = (float*)malloc(nnzB * sizeof(float));

  // Generate random sparse matrices
  srand(0);
  for (int i = 0; i <= m; i++) {
    csrRowPtrA[i] = i * nnzA / m;
  }
  for (int i = 0; i < nnzA; i++) {
    csrColIndA[i] = rand() % n;
    csrValA[i] = (float)rand() / RAND_MAX;
  }
  for (int i = 0; i <= n; i++) {
    csrRowPtrB[i] = i * nnzB / n;
  }
  for (int i = 0; i < nnzB; i++) {
    csrColIndB[i] = rand() % n;
    csrValB[i] = (float)rand() / RAND_MAX;
  }

  CHECK_CUDA(hipEventRecord(start));

  // Allocate device memory
  int *d_csrRowPtrA, *d_csrColIndA;
  float *d_csrValA;
  int *d_csrRowPtrB, *d_csrColIndB;
  float *d_csrValB;
  int *d_csrRowPtrC;
  int *d_csrColIndC;
  float *d_csrValC;

  CHECK_CUDA(hipMalloc((void**)&d_csrRowPtrA, (m + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_csrColIndA, nnzA * sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_csrValA, nnzA * sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&d_csrRowPtrB, (n + 1) * sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_csrColIndB, nnzB * sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_csrValB, nnzB * sizeof(float)));

  // Copy data from host to device
  CHECK_CUDA(hipMemcpy(d_csrRowPtrA, csrRowPtrA, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrColIndA, csrColIndA, nnzA * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrValA, csrValA, nnzA * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrRowPtrB, csrRowPtrB, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrColIndB, csrColIndB, nnzB * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_csrValB, csrValB, nnzB * sizeof(float), hipMemcpyHostToDevice));

  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));
  float initTime;
  CHECK_CUDA(hipEventElapsedTime(&initTime, start, stop));
  initTime /= 1000.0;  // Convert to seconds

  CHECK_CUDA(hipEventRecord(start));

  // Initialize CuSPARSE
  hipsparseHandle_t handle;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Create matrix descriptors
  hipsparseMatDescr_t descr_A, descr_B, descr_C;
  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr_A));
  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr_B));
  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr_C));

  // Perform matrix multiplication
  int nnzC;
  CHECK_CUSPARSE(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));
  CHECK_CUSPARSE(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        m, n, n, descr_A, nnzA, d_csrRowPtrA, d_csrColIndA,
        descr_B, nnzB, d_csrRowPtrB, d_csrColIndB,
        descr_C, d_csrRowPtrC, &nnzC));

  CHECK_CUDA(hipMalloc((void**)&d_csrColIndC, nnzC * sizeof(int)));
  CHECK_CUDA(hipMalloc((void**)&d_csrValC, nnzC * sizeof(float)));

  float alpha = 1.0f;
  float beta = 0.0f;
  CHECK_CUSPARSE(hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        m, n, n, &alpha, descr_A, nnzA, d_csrValA, d_csrRowPtrA, d_csrColIndA,
        descr_B, nnzB, d_csrValB, d_csrRowPtrB, d_csrColIndB,
        &beta, descr_C, d_csrValC, d_csrRowPtrC, d_csrColIndC));

  CHECK_CUDA(hipEventRecord(stop));
  CHECK_CUDA(hipEventSynchronize(stop));
  float multTime;
  CHECK_CUDA(hipEventElapsedTime(&multTime, start, stop));
  multTime /= 1000.0;  // Convert to seconds

  float totalTime = initTime + multTime;

  // Copy result from device to host
  csrRowPtrC = (int*)malloc((m + 1) * sizeof(int));
  csrColIndC = (int*)malloc(nnzC * sizeof(int));
  csrValC = (float*)malloc(nnzC * sizeof(float));

  CHECK_CUDA(hipMemcpy(csrRowPtrC, d_csrRowPtrC, (m + 1) * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(csrColIndC, d_csrColIndC, nnzC * sizeof(int), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(csrValC, d_csrValC, nnzC * sizeof(float), hipMemcpyDeviceToHost));

  // Print profiling results
  size_t memUsage = ((m + 1) + nnzA + nnzA) * sizeof(int) + nnzA * sizeof(float) +
    ((n + 1) + nnzB + nnzB) * sizeof(int) + nnzB * sizeof(float) +
    ((m + 1) + nnzC + nnzC) * sizeof(int) + nnzC * sizeof(float);

  printProfiling(m, n, nnzA, nnzB, nnzC, initTime, multTime, totalTime, memUsage);

  // Clean up
  CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_A));
  CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_B));
  CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr_C));
  CHECK_CUSPARSE(hipsparseDestroy(handle));

  CHECK_CUDA(hipFree(d_csrRowPtrA));
  CHECK_CUDA(hipFree(d_csrColIndA));
  CHECK_CUDA(hipFree(d_csrValA));
  CHECK_CUDA(hipFree(d_csrRowPtrB));
  CHECK_CUDA(hipFree(d_csrColIndB));
  CHECK_CUDA(hipFree(d_csrValB));
  CHECK_CUDA(hipFree(d_csrRowPtrC));
  CHECK_CUDA(hipFree(d_csrColIndC));
  CHECK_CUDA(hipFree(d_csrValC));

  free(csrRowPtrA);
  free(csrColIndA);
  free(csrValA);
  free(csrRowPtrB);
  free(csrColIndB);
  free(csrValB);
  free(csrRowPtrC);
  free(csrColIndC);
  free(csrValC);

  return 0;
}
